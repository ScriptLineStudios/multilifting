#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include <stdint.h>
#include <assert.h>
#include <math.h>
#include <stdbool.h>

#define MAX(x, y) (((x) > (y)) ? (x) : (y))

#include "cubiomes/rng.h"
#include "cubiomes/finders.h"

typedef enum {
    STRUCTURE, BIOME, PILLAR
} ConstraintType;

typedef struct {
    ConstraintType tp; 

    int type;
    int x, z;
    
    int chunk_range, offset_x, offset_z, reg_x, reg_z;
    uint64_t salt;
} Constraint;

#define STRUCTURE_CONSTRAINT(t, _x, _z) (Constraint){.tp=STRUCTURE, .type=t, .x=_x, .z=_z}
#define BIOME_CONSTRAINT(t, _x, _z) (Constraint){.tp=BIOME, .type=t, .x=_x, .z=_z}
#define PILLAR_CONSTRAINT(seed) (Constraint){.tp=PILLAR, .type=seed}

typedef struct {
    Constraint *cons;
    size_t length;
} Constraints;

void parse_constraints(Constraints constraints) {
    if (constraints.length < 1) {
        fprintf(stderr, "ERROR: You need at least one constraint!");
        exit(1);
    }

    StructureConfig conf;
    int total_bits = 0;
    for (size_t i = 0; i < constraints.length; i++) {
        Constraint *c = &constraints.cons[i];
        if (c->tp != STRUCTURE) { // only structure constraints need to be parsed :D
            continue;
        }
 
        getStructureConfig(c->type, MC_1_16, &conf);
        c->salt = conf.salt;
        c->chunk_range = conf.chunkRange;
        
        int cx = c->x >> 4;
        int cz = c->z >> 4;

        int _cx = cx < 0 ? cx - conf.regionSize + 1 : cx;
        int _cz = cz < 0 ? cz - conf.regionSize + 1 : cz;

        c->reg_x = floor(_cx / conf.regionSize);
        c->reg_z = floor(_cz / conf.regionSize);

        c->offset_x = cx - (c->reg_x * conf.regionSize);
        c->offset_z = cz - (c->reg_z * conf.regionSize);

        total_bits += (int)log2((conf.chunkRange) & (-conf.chunkRange)) * 2;
    }
    if (total_bits < 20) {
        fprintf(stderr, "WARNING: Not enough bits to narrow down structure seed! (%d/%d)\n", total_bits, 20);
        // exit(1);
    }
}

uint64_t gcd(uint64_t a, uint64_t b) {
    if (a == 0)
        return b;
    if (b == 0)
        return a;
    if (a == b)
        return a;
    if (a > b)
        return gcd(a - b, b);
    return gcd(a, b - a);
}

__device__ __host__ void goBack2(uint64_t* rand) {
    *rand = (*rand * 254681119335897ULL + 120305458776662ULL) & MASK48;
}

__device__ __host__ void goBack(uint64_t* rand) {
    *rand = (*rand * 246154705703781ULL + 107048004364969ULL) & MASK48;
}

// stolen from https://github.com/michel-leonard/C-MathSnip/blob/main/mod_inv.c :)
uint64_t modinv(uint64_t ra, uint64_t rb) {
    uint64_t rc, sa = 1, sb = 0, sc, i = 0;
    if (rb > 1) do {
            rc = ra % rb;
            sc = sa - (ra / rb) * sb;
            sa = sb, sb = sc;
            ra = rb, rb = rc;
        } while (++i, rc);
    sa *= (i *= ra == 1) != 0;
    sa += (i & 1) * sb;
    return sa;
}

typedef struct {
    uint64_t *seeds;
    size_t size;
} Seeds;

Seeds solve_constraints_for_lower20(Constraints constraints) {
    int max = 0;
    for (size_t i = 0; i < constraints.length; i++) {
        if ((int)log2(constraints.cons[i].chunk_range & -constraints.cons[i].chunk_range)) {
            max = MAX(max, constraints.cons[i].chunk_range);
        }
    }
    int mp = max & -max; 
    int lp = (int)log2(mp);

    size_t num_lower = 0;
    uint64_t *valid_lower = (uint64_t *)malloc(sizeof(uint64_t) * (int)((100.0/100.0) * (1<<(17 + lp))));
    // huge thank you to Kris for his great writeup on bitlifing: https://github.com/Kludwisz/BitLifting/
    for (uint64_t lower = 0; lower < 1ull<<(17 + lp); lower++) {
        for (size_t i = 0; i < constraints.length; i++) {
            Constraint c = constraints.cons[i];
            if (c.tp != STRUCTURE) {
                continue;
            }
            int p = c.chunk_range & (-c.chunk_range);
            if (p < 2) {
                continue;
            }
            uint64_t lower_seed = ((uint64_t)c.reg_x*341873128712ULL + (uint64_t)c.reg_z*132897987541ULL + lower + (uint64_t)c.salt);
            setSeed(&lower_seed, lower_seed);
            if (nextInt(&lower_seed, c.chunk_range) % p != c.offset_x % p || nextInt(&lower_seed, c.chunk_range) % p != c.offset_z % p) {
                goto next_lower;
            }
        }
        valid_lower[num_lower] = lower;
        num_lower++;
next_lower:
        ;
    }

    return (Seeds){.seeds=valid_lower, .size=num_lower};
}

#define LENGTH(x) (sizeof(x) / sizeof(x[0]))
__device__ __managed__ unsigned long long int checked = 0;
__device__ __managed__ unsigned long long int buffer_size = 0;

__global__ void find_structure_seeds(Constraint best_constraint, Constraints constraints, uint64_t region_seed, uint64_t basis, uint64_t end, uint64_t reduced_mod, uint64_t *buffer) {
    uint64_t input_seed = blockDim.x * blockIdx.x + threadIdx.x;
    if (input_seed % reduced_mod != basis || input_seed > end) {
        return;
    } 
    uint64_t full_region_seed = (input_seed << 20) | region_seed; 
    goBack2(&full_region_seed);
    int x = nextInt(&full_region_seed, best_constraint.chunk_range);
    int z = nextInt(&full_region_seed, best_constraint.chunk_range);
    if (x != best_constraint.offset_x || z != best_constraint.offset_z) { // the second part should never really happen, just a sanity check...
        return;
    }
    goBack2(&full_region_seed);

    full_region_seed ^= 0x5deece66d;
    uint64_t structure_seed = (full_region_seed - best_constraint.reg_x * 341873128712ULL - best_constraint.reg_z * 132897987541ULL - (uint64_t)best_constraint.salt) & MASK48;
    for (size_t i = 0; i < constraints.length; i++) {
        Constraint *c = &constraints.cons[i];

        if (c->tp == PILLAR) {
            uint64_t ss = structure_seed;
            if ((nextLong(&ss) & 65535) != (uint64_t)c->type) {
                return;
            }
        }
        if (c->tp != STRUCTURE) {
            continue;
        }

        uint64_t ss = structure_seed;
        setSeed(&ss, c->reg_x*341873128712ull + c->reg_z*132897987541ull + ss + (uint64_t)c->salt);
        if (nextInt(&ss, c->chunk_range) != c->offset_x || nextInt(&ss, c->chunk_range) != c->offset_z) {
            return;
        }
    }
    // atomicAdd(&checked, 1ull);
    uint64_t r = atomicAdd(&buffer_size, 1ull);
    buffer[r] = structure_seed;
}

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
    exit(code);
  }
}

Constraints make_gpu_constraints(Constraints cpu_constraints) {
    Constraint *new_constraints;
    hipMalloc((void **)&new_constraints, sizeof(Constraint) * cpu_constraints.length);
    hipMemcpy(new_constraints, cpu_constraints.cons, sizeof(Constraint) * cpu_constraints.length, hipMemcpyHostToDevice); 
    return (Constraints){.cons=new_constraints, .length=cpu_constraints.length};
}

int main(void) {
    Constraint constraints[] = {
        STRUCTURE_CONSTRAINT(Village, -16 + 16, 736),
        STRUCTURE_CONSTRAINT(Igloo, 272, 608),
    };

    hipSetDevice(0);

    Constraints cons = (Constraints){.cons=constraints, .length=LENGTH(constraints)};
    parse_constraints(cons);

    Seeds lower20_seeds = solve_constraints_for_lower20(cons);
    printf("Found %ld lower20 seeds\n", lower20_seeds.size);

    int max = 0;
    int best_index = -1;
    for (size_t i = 0; i < cons.length; i++) {
        Constraint *c = &cons.cons[i];
        uint64_t param = c->chunk_range / gcd(8, c->chunk_range);
        if (param > (uint64_t)max) {
            best_index = i;
            max = (int)param;
        }
    }
    Constraint best_constraint = cons.cons[best_index];

    Constraints gpu_cons = make_gpu_constraints(cons);

    FILE *seed_file = fopen("structure_seeds.txt", "w");
    for (size_t i = 0; i < lower20_seeds.size; i++) {
        uint64_t lower20 = lower20_seeds.seeds[i];
        uint64_t region_lower20 = (((uint64_t)best_constraint.reg_x*341873128712ULL + (uint64_t)best_constraint.reg_z*132897987541ULL + lower20 + (uint64_t)best_constraint.salt)) & 0xFFFFF;
        setSeed(&region_lower20, region_lower20);
        int x = nextInt(&region_lower20, best_constraint.chunk_range);
        int z = nextInt(&region_lower20, best_constraint.chunk_range);
        (void)x;
        (void)z;
        region_lower20 = region_lower20 & 0xFFFFF; 
        uint64_t L3 = (region_lower20 >> 17) & 0x7;
        uint64_t C = best_constraint.chunk_range;
        uint64_t Z = best_constraint.offset_z;
        uint64_t d = gcd(8, C);
        uint64_t reduced_mod = C / d;
        uint64_t inv = modinv(8 / d, C / d);
        uint64_t basis = inv * ((Z - L3) / d) % reduced_mod;
        uint64_t end = (1ull<<28ull) - 1ull;

        uint64_t *seed_buffer;
        GPU_ASSERT(hipMallocManaged((void **)&seed_buffer, sizeof(uint64_t) * 4000000));
        
        find_structure_seeds<<<2097152, 128>>>(best_constraint, gpu_cons, region_lower20, basis, end, reduced_mod, seed_buffer);
        GPU_ASSERT(hipDeviceSynchronize()); 
        printf("%ld/%ld => %lld buffer_size: %lld\n", i, lower20_seeds.size, checked, buffer_size);

        for (int i = 0; i < buffer_size; i++) {
            fprintf(seed_file, "%lu\n", seed_buffer[i]);
        }

        buffer_size = 0;
        hipFree(seed_buffer);
    }
    printf("Found: %lld structure seeds!\n", checked);

    fclose(seed_file);
    free(lower20_seeds.seeds);
    return 0;
}